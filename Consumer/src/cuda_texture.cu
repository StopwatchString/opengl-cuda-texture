// kernel.cu
#include <hip/hip_runtime.h>

__global__ void helloKernel(char* str) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Change 'o' to '0'
    if (str[idx] == 'o') {
        str[idx] = '0';
    }
}

// Function declaration that will be called from C++
extern "C" void launchHelloKernel(char* str, int size);

// Implementation of the function that launches the kernel
void launchHelloKernel(char* str, int size) {
    char* d_str;
    hipMalloc((void**)&d_str, size);
    hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

    helloKernel<<<1, size >>>(d_str);

    hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
    hipFree(d_str);
}